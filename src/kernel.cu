
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <stdio.h>
#include <string.h>

__device__ __constant__ float D_H[ 3*3 ];

__device__ float norm(float val, int length) {
    float mean = length/2;
    float std = length/2;
    return (val-mean)/std;
}

__device__ float unorm(float val, int length) {
    float mean = length/2;
    float std = length/2;
    return val*std + mean;
}

__device__ void projectedCoord(int x, int y, int *xp, int *yp, int xlen, int ylen) {

    //printf("%d, %d \n", x, y);
    //NORMALIZE INPUT
    float nx = norm(x,xlen);
    float ny = norm(y,ylen);

    //printf("%f, %f \n", nx, ny);
    int sH = 3;
    float w = 1; //Assume that the projection starts from y=1


    float hx = nx*D_H[ sH*0+0] + ny*D_H[ sH*0+1 ] + w*D_H[ sH*0+2 ];
    float hy = nx*D_H[ sH*1+0] + ny*D_H[ sH*1+1 ] + w*D_H[ sH*1+2 ];
    float hw = nx*D_H[ sH*2+0] + ny*D_H[ sH*2+1 ] + w*D_H[ sH*2+2 ];

    //printf("%f, %f, %f\n",D_H[ sH*0+0], D_H[ sH*0+1], D_H[ sH*0+2]);
    //printf("%f, %f, %f\n",D_H[ sH*1+0], D_H[ sH*1+1], D_H[ sH*1+2]);
    //printf("%f, %f, %f\n",D_H[ sH*2+0], D_H[ sH*2+1], D_H[ sH*2+2]);
    //printf("%f %f %f \n", hx, hy, hw);

    //Unormalize Output
    *xp = unorm(hx/hw, xlen);
    *yp = unorm(hy/hw, ylen);

    //printf("%d, %d \n", *xp, *yp);

}

__device__ int im_idx(int r, int c, int width, int channels) {
    return channels*(width*r+c);
}

__device__ bool val_rc(int r, int c, int width, int height) {
    return r>=0 && r<height && c>=0 && c<width;
}

__global__ void proj_sub_tresh(unsigned char* img0, unsigned char* img1, unsigned char* out_img, int Width, int Height) {
    const unsigned int c = ( (blockDim.y * blockIdx.y) + threadIdx.y );
    const unsigned int r = ( (blockDim.x * blockIdx.x) + threadIdx.x );

    const unsigned int treshold = 60;
    const unsigned int ch = 3; //Channel
    const unsigned int s = sizeof(unsigned char);
    const unsigned int W = Width;
    int o_img_idx;
    int i_img_idx;
    unsigned int subval, subval0, subval1, subval2;
    int rp;
    int cp;
    //Projection, Background Sub, Treshold

    // Not sure why I wrote the matrix in this manner where the r column is reversed using x,y notation
    // Need to look deeper into and be fixed
    projectedCoord(c,r,  &cp, &rp, Width, Height);

    //printf("%d, %d \n", rp, cp);
    if ( val_rc(rp,cp, Width, Height) && val_rc(r,c, Width, Height) ) {


        o_img_idx = im_idx(r,c,  Width,ch);
        i_img_idx = im_idx(rp,cp,  Width,ch);
        subval0 = abs( img1[ o_img_idx+0 ] - img0[ i_img_idx+0 ] );
        subval1 = abs( img1[ o_img_idx+1 ] - img0[ i_img_idx+1 ] );
        subval2 = abs( img1[ o_img_idx+2 ] - img0[ i_img_idx+2 ] );
        subval = .21265*subval0 + .7152*subval1 + .0722*subval2;

        if (subval > treshold) {
            out_img[ o_img_idx+0 ] = subval;
            out_img[ o_img_idx+1 ] = subval;
            out_img[ o_img_idx+2 ] = subval;
        }
        //out_img[ o_img_idx+0 ] = img0[ i_img_idx+0];
        //out_img[ o_img_idx+1 ] = img0[ i_img_idx+1];
        //out_img[ o_img_idx+2 ] = img0[ i_img_idx+2];


    }
}
