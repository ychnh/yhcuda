#include "hip/hip_runtime.h"
/*
This is the central piece of code. This file implements a class
 that takes data in on the cpu side, copies
it to the gpu, and exposes functions (increment and retreive) that let
you perform actions with the GPU

This class will get translated into python via swig
*/

#include <kernel.cu>
#include <manager.hh>
#include <assert.h>
#include <iostream>
#include <cmath>
#include <stdio.h>
#include <string.h>

using namespace std;

int img_s(int width, int height, int chan, int size) {
    return width*height*chan*size;
}

void mov_obj_detect( unsigned char* img0, unsigned char* img1, unsigned char* out_img, float* H_filter, int W, int H) {
    hipMemcpyToSymbol(HIP_SYMBOL( D_H), H_filter, 3*3*sizeof(float) );
    // Alloc Cpy Var
    unsigned int s = sizeof(unsigned char);
    unsigned int c = 3;

    unsigned char* D_img0;
    hipMalloc( (void **) &D_img0, img_s(W,H,c,s) );
    hipMemcpy( D_img0, img0, img_s(W,H,c,s), hipMemcpyHostToDevice);

    unsigned char* D_img1;
    hipMalloc( (void **) &D_img1, img_s(W,H,c,s) );
    hipMemcpy( D_img1, img1, img_s(W,H,c,s), hipMemcpyHostToDevice);

    unsigned char* D_out_img;
    hipMalloc( (void **) &D_out_img, img_s(W,H,c,s));

    // Call Kernel
    unsigned int n = 32; // Block_Size
    unsigned int N = ceil( ((double) H) /n); // Grid Rows
    unsigned int M = ceil( ((double) W)/n); // Grid Cols

    dim3 gridDims(N,M,1);
    dim3 blockDims(n,n,1);
    proj_sub_tresh<<< gridDims, blockDims >>>(D_img0, D_img1, D_out_img, W, H);
    hipMemcpy( out_img, D_out_img, img_s(W,H,c,s), hipMemcpyDeviceToHost);

    // Free Var
    hipFree(D_img0);
    hipFree(D_img1);
    hipFree(D_out_img);
}


